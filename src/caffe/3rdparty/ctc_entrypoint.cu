#include <cstddef>
#include <iostream>
#include <algorithm>

#define __HIPCC__
#include "caffe/3rdparty/ctc.h"
#include "caffe/3rdparty/detail/cpu_ctc.cuh"
#include "caffe/3rdparty/detail/gpu_ctc.cuh"


extern "C" {

ctcStatus_t compute_ctc_loss_gpu(const float* const activations,
                                 float* gradients,
                                 const int* const flat_labels,
                                 const int* const label_lengths,
                                 const int* const input_lengths,
                                 int alphabet_size,
                                 int minibatch,
                                 float *costs,
                                 void *workspace,
                                 ctcOptions options) {
        GpuCTC<float> ctc(alphabet_size, minibatch, workspace, options.stream,
                          options.blank_label);

        if (gradients != NULL)
            return ctc.cost_and_grad(activations, gradients, costs,
                                     flat_labels, label_lengths,
                                     input_lengths);
        else
            return ctc.score_forward(activations, costs, flat_labels,
                                     label_lengths, input_lengths);
        return CTC_STATUS_EXECUTION_FAILED;
}
}
